// #include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include<sys/time.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. A 2D thread block and 2D grid are used. sumArraysOnHost sequentially
 * iterates through vector elements on the host.
 */

using T=int;

double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec+(double)tp.tv_usec*1.e-6);
}

void initialData(T *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (T)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumMatrixOnHost(T *A, T *B, T *C, const int nx,
                     const int ny)
{
    T *ia = A;
    T *ib = B;
    T *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];

        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}


void checkResult(T *hostRef, T *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %d gpu %d\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

// grid 2D block 2D
__global__ void sumMatrixOnGPU2D(T *MatA, T *MatB, T *MatC, int nx,
                                 int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    (hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    (hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(T);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    T *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (T *)malloc(nBytes);
    h_B = (T *)malloc(nBytes);
    hostRef = (T *)malloc(nBytes);
    gpuRef = (T *)malloc(nBytes);

    // initialize data at host side
    double iStart = cpuSecond();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    double iElaps = cpuSecond() - iStart;
    printf("Matrix initialization elapsed %f sec\n", iElaps);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = cpuSecond();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = cpuSecond() - iStart;
    printf("sumMatrixOnHost elapsed %f sec\n", iElaps);

    // malloc device global memory
    T *d_MatA, *d_MatB, *d_MatC;
    (hipMalloc((void **)&d_MatA, nBytes));
    (hipMalloc((void **)&d_MatB, nBytes));
    (hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    (hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    (hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    iStart = cpuSecond();
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    (hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
           grid.y,
           block.x, block.y, iElaps);
    // check kernel error
    (hipGetLastError());

    // copy kernel result back to host side
    (hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    (hipFree(d_MatA));
    (hipFree(d_MatB));
    (hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    (hipDeviceReset());

    return (0);
}
